#include "hip/hip_runtime.h"
/*该版本代码的选择器与CPU略有不同，是尽可能少产生异常值的标准*/

#include "alp_gpu.hpp"
#include <hip/hip_runtime.h>
#include <cassert>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <stdexcept>
#include <iostream>
#include <climits> // LLONG_MAX, LLONG_MIN

using std::uint8_t; using std::uint32_t; using std::uint64_t;

namespace alp_gpu {

// ===================== 常量（与 CPU 版一致） =====================
// e,f 取值范围 0..18（19个）
__device__ __constant__ double D_EXP_ARR[19] = {
  1.0, 10.0, 100.0, 1000.0, 10000.0, 100000.0, 1000000.0, 10000000.0,
  100000000.0, 1000000000.0, 10000000000.0, 100000000000.0, 1000000000000.0,
  10000000000000.0, 100000000000000.0, 1000000000000000.0,
  10000000000000000.0, 100000000000000000.0, 1000000000000000000.0
};
__device__ __constant__ double D_FRAC_ARR[20] = {
  1.0, 0.1, 0.01, 0.001, 0.0001, 0.00001, 0.000001, 0.0000001,
  0.00000001, 0.000000001, 0.0000000001, 0.00000000001, 0.000000000001,
  0.0000000000001, 0.00000000000001, 0.000000000000001, 0.0000000000000001,
  0.00000000000000001, 0.000000000000000001
};
// ALPrd：3bit 字典（大小 8），CUTTING_LIMIT = 16（与 CPU 保持）
static constexpr int   DICT_BW  = 3;
static constexpr int   DICT_SZ  = 1 << DICT_BW;
static constexpr int   CUT_LIM  = 16;
static constexpr int   MAX_VEC  = 4096; // 保护上限（<= 4096）

// ===================== 设备端位流 Writer/Reader =====================
struct BitWriter {
    uint8_t* buf;      // 全局输出缓冲（按 bit 偏移写）
    uint64_t bitpos;   // 写入起始 bit 偏移（每块各自独立）
    // __device__ void put1(int b){
    //     if (!b) { ++bitpos; return; } // 初始缓冲区已清零；只在写 1 时 OR 即可
    //     uint64_t byte = bitpos >> 3;
    //     int      off  = 7 - int(bitpos & 7ULL);
    //     unsigned int* p32 = reinterpret_cast<unsigned int*>(&buf[byte & ~3ULL]);
    //     unsigned int  m   = 1u << ((byte & 3ULL)*8 + off);
    //     atomicOr(p32, m);
    //     ++bitpos;
    // }
    // __device__ void putN(uint64_t v, int bits){
    //     for(int i=bits-1;i>=0;--i) put1( (v>>i) & 1ULL );
    // }
    __device__ void put1(int b){
        if (!b) { ++bitpos; return; }          // 初始缓冲区已清零
        uint64_t byte = bitpos >> 3;
        int      off  = 7 - int(bitpos & 7ULL);
        buf[byte] = uint8_t(buf[byte] | (uint8_t(1u) << off));  // 普通字节写
        ++bitpos;
    }
    __device__ void putN(uint64_t v, int bits){
        for(int i=bits-1;i>=0;--i) put1( (v>>i) & 1ULL );
    }
};
struct BitReader {
    const uint8_t* buf;
    uint64_t bitpos;
    __device__ int get1(){
        uint64_t byte = bitpos >> 3;
        int off = 7 - int(bitpos & 7ULL);
        int b = (buf[byte] >> off) & 1;
        ++bitpos; return b;
    }
    __device__ uint64_t getN(int bits){
        uint64_t v=0;
        for(int i=0;i<bits;++i){ v = (v<<1) | get1(); }
        return v;
    }
};

// ===================== 公共工具（GPU） =====================
__device__ __forceinline__ int width_needed_unsigned(unsigned long long range){
    if (range==0ULL) return 1;
    int c=0; while(range){ ++c; range>>=1ULL; } return c;
}

__device__ inline long long fast_round_double(double x){
    // 与 CPU 版相同的“甜 spot”整型回转
    const double SWEET = double((1ULL<<51) + (1ULL<<52));
    return (long long)(x + SWEET) - (long long)SWEET;
}

// ============= 采样 & 模式判定（与 CPU 逻辑对齐） =============
template<typename T>
__device__ inline bool alp_exact_equal(T v, uint8_t e, uint8_t f){
    if constexpr (std::is_same_v<T,double>) {
        double enc = v * D_EXP_ARR[e] * D_FRAC_ARR[f];
        long long I = fast_round_double(enc);
        // 解码：I * 10^{f} * 10^{-e} = I * (1/FRAC[f]) * FRAC[e]
        double dec = double(I) * (1.0 / D_FRAC_ARR[f]) * D_FRAC_ARR[e];
        return dec==v;
    } else {
        float enc = v * float(D_EXP_ARR[e]) * float(D_FRAC_ARR[f]);
        int   I   = __float2int_rn(enc);
        float dec = float(I) * (1.0f/float(D_FRAC_ARR[f])) * float(D_FRAC_ARR[e]);
        return dec==v;
    }
}

template<typename T>
__device__ inline bool is_high_precision_value(T v){
    for(uint8_t e=0;e<=18;++e){
        for(uint8_t f=0; f<=e; ++f){
            if (alp_exact_equal<T>(v,e,f)) return false;
        }
    }
    return true;
}

// 估计 ALP 异常率（对采样子集）
template<typename T>
__device__ inline double estimate_alp_exception_rate(const T* a, int n){
    int best_exc = n+1;
    for(uint8_t e=0;e<=18;++e){
        for(uint8_t f=0;f<=e;++f){
            int exc=0; long long mn=LLONG_MAX, mx=LLONG_MIN;
            for(int i=0;i<n;++i){
                double enc = double(a[i]) * D_EXP_ARR[e] * D_FRAC_ARR[f];
                long long I = fast_round_double(enc);
                double dec = double(I) * (1.0/double(D_FRAC_ARR[f])) * D_FRAC_ARR[e];
                if (dec==double(a[i])) { mn = (mn<I?mn:I); mx = (mx>I?mx:I); }
                else ++exc;
            }
            if (exc < best_exc) best_exc = exc;
        }
    }
    return double(best_exc)/double(n);
}

template<typename T>
__device__ inline CompressionMode decide_mode_block(const T* blk, int n, int sample_stride){
    int total=0, highp=0;
    for(int i=0;i<n; i+=sample_stride){ ++total; if (is_high_precision_value<T>(blk[i])) ++highp; }
    if (total>0 && (double)highp/total > 0.5) return CompressionMode::ALPrd;

    const int SAMPLE_MAX = (32 < total ? 32 : total);
    if (SAMPLE_MAX<=0) return CompressionMode::ALP;
    T tmp[32];
    int k=0;
    for(int i=0;i<n && k<SAMPLE_MAX; i+=sample_stride) tmp[k++]=blk[i];
    double er = estimate_alp_exception_rate<T>(tmp, k);
    if (er>0.2) return CompressionMode::ALPrd;
    return CompressionMode::ALP;
}

// ============= ALP 单向量：统计 & 写入 =============
template<typename T>
__device__ inline void alp_vector_analyze(const T* v, int n, uint8_t e, uint8_t f,
                                          short& bitw, long long& FOR,
                                          int& exc_cnt){
    long long mn=LLONG_MAX, mx=LLONG_MIN;
    exc_cnt=0;
    for(int i=0;i<n;++i){
        double enc = double(v[i]) * D_EXP_ARR[e] * D_FRAC_ARR[f];
        long long I = fast_round_double(enc);
        double dec = double(I) * (1.0/double(D_FRAC_ARR[f])) * D_FRAC_ARR[e];
        if (dec==double(v[i])) { mn=(mn<I?mn:I); mx=(mx>I?mx:I); }
        else ++exc_cnt;
    }
    unsigned long long range = (mn==LLONG_MAX)? 0ULL : (unsigned long long)(mx - mn);
    bitw = (short)width_needed_unsigned(range);
    FOR  = (mn==LLONG_MAX?0:mn);
}

template<typename T>
__device__ inline uint64_t alp_vector_size_bits(int n, uint8_t e, uint8_t f,
                                                short bitw, int exc_cnt){
    int val_bits = std::is_same_v<T,double> ? 64 : 32;
    return 1 + 8+8+16+64+32 + uint64_t(n)*bitw + 16 + uint64_t(exc_cnt)*(val_bits+16);
}

template<typename T>
__device__ inline void alp_vector_write(BitWriter& bw, const T* v, int n,
                                       uint8_t e, uint8_t f, short bitw, long long FOR){
    assert(n <= MAX_VEC);
    bw.put1(1); // useALP = 1
    bw.putN((uint64_t)e, 8); bw.putN((uint64_t)f, 8);
    bw.putN((uint64_t)bitw, 16);
    bw.putN((uint64_t)FOR, 64);
    bw.putN((uint64_t)n, 32);

    int exc_cnt=0;
    int      exc_pos[MAX_VEC];
    uint64_t exc_val[MAX_VEC];

    for(int i=0;i<n;++i){
        double enc = double(v[i]) * D_EXP_ARR[e] * D_FRAC_ARR[f];
        long long I = fast_round_double(enc);
        double dec = double(I) * (1.0/double(D_FRAC_ARR[f])) * D_FRAC_ARR[e];
        if (dec==double(v[i])) {
            uint64_t packed = (uint64_t)(I - FOR);
            bw.putN(packed, bitw);
        } else {
            bw.putN(0, bitw); // 占位
            if constexpr (std::is_same_v<T,double>) {
                uint64_t raw = *reinterpret_cast<const uint64_t*>(&v[i]);
                exc_val[exc_cnt] = raw;
            } else {
                uint32_t raw = *reinterpret_cast<const uint32_t*>(&v[i]);
                exc_val[exc_cnt] = raw;
            }
            exc_pos[exc_cnt] = i;
            ++exc_cnt;
        }
    }
    bw.putN((uint64_t)exc_cnt, 16);
    for(int k=0;k<exc_cnt;++k){
        if constexpr (std::is_same_v<T,double>) bw.putN(exc_val[k], 64);
        else bw.putN(exc_val[k], 32);
        bw.putN((uint64_t)exc_pos[k], 16);
    }
}

// ============= ALPrd：字典与写入（精确 top-8，本地小表，无 shared） =============
template<typename T> struct ALPrdDict {
    uint8_t rightBW;        // 右半位宽
    uint8_t leftBW;         // 左半位宽
    uint32_t dict[DICT_SZ]; // left parts 字典（<= 2^leftBW-1）
};

__device__ inline uint32_t mask_lo(int bits){
    return (bits >= 32) ? 0xFFFFFFFFu : ((1u<<bits) - 1u);
}

template<typename T>
__device__ inline void alprd_find_best(const uint64_t* in, int n, ALPrdDict<T>& D){
    // 穷举左宽 1..CUT_LIM，精确统计频次 → 取 top-8
    double best_score = 1e100; int best_rbw = int(sizeof(T)*8) - 1;
    uint32_t best_dict[DICT_SZ] = {0};

    for(int lbw=1; lbw<=CUT_LIM; ++lbw){
        int rbw = int(sizeof(T)*8) - lbw;
        uint32_t lmask = mask_lo(lbw);

        // 频次统计（最多 n=vectorSize 个不同 left），用小表（最多 n 项）
        uint32_t uniq_left[MAX_VEC]; int cnt[MAX_VEC];
        int u = 0;

        for(int i=0;i<n;++i){
            uint32_t left = (uint32_t)((in[i] >> rbw) & lmask);
            int j=0; for(; j<u; ++j) if (uniq_left[j]==left) { ++cnt[j]; break; }
            if (j==u){ uniq_left[u]=left; cnt[u]=1; ++u; }
        }
        // 选 top-8
        uint32_t dict[DICT_SZ]={0};
        int used = (DICT_SZ < u ? DICT_SZ : u);
        for(int k=0;k<used;++k){
            int best=-1, id=-1;
            for(int j=0;j<u;++j){
                bool taken=false;
                for(int t=0;t<k;++t) if (dict[t]==uniq_left[j]) { taken=true; break; }
                if (taken) continue;
                if (cnt[j]>best){ best=cnt[j]; id=j; }
            }
            dict[k] = uniq_left[id];
        }
        // 异常数量
        int keep=0;
        for(int k=0;k<used;++k){
            for(int j=0;j<u;++j) if (uniq_left[j]==dict[k]) { keep += cnt[j]; break; }
        }
        int exc = n - keep;

        // 估计位数：n*(DICT_BW+rbw) + dict(8*lbw) + 异常(16+lbw)*exc + 头(1+32+8)
        double bits = 1 + 32 + 8
                      + double(n)*(DICT_BW + rbw)
                      + double(DICT_SZ)*lbw
                      + 16.0*exc + double(lbw)*exc;

        if (bits < best_score){
            best_score = bits;
            best_rbw   = rbw;
            for(int k=0;k<DICT_SZ;++k) best_dict[k]=dict[k];
        }
    }
    D.rightBW = (uint8_t)best_rbw;
    D.leftBW  = (uint8_t)(int(sizeof(T)*8) - best_rbw);
    for(int k=0;k<DICT_SZ;++k) D.dict[k]=best_dict[k];
}

template<typename T>
__device__ inline uint64_t alprd_vector_size_bits(int n, const ALPrdDict<T>& D, int exc_cnt){
    return 1 + 32 + 8 + uint64_t(n)*(DICT_BW + D.rightBW)
           + DICT_SZ*D.leftBW + 16 + uint64_t(exc_cnt)*(D.leftBW+16);
}

template<typename T>
__device__ inline void alprd_vector_write(BitWriter& bw, const uint64_t* in, int n,
                                          const ALPrdDict<T>& D){
    assert(n <= MAX_VEC);
    bw.put1(0); // useALP=0
    bw.putN((uint64_t)n, 32);
    bw.putN((uint64_t)D.rightBW, 8);

    // 逐值输出：leftIdx(3)+right(rbw)，同时记录异常
    int exc_cnt=0; uint16_t exc_pos[MAX_VEC]; uint32_t exc_left[MAX_VEC];
    uint64_t right_mask = (D.rightBW==64)? ~0ULL : ((1ULL<<D.rightBW)-1ULL);
    uint32_t left_mask  = mask_lo(D.leftBW);

    for(int i=0;i<n;++i){
        uint64_t right = in[i] & right_mask;
        uint32_t left  = (uint32_t)((in[i] >> D.rightBW) & left_mask);
        short idx = DICT_SZ;
        for(int k=0;k<DICT_SZ;++k){ if (D.dict[k]==left){ idx=(short)k; break; } }
        if (idx<DICT_SZ){
            bw.putN((uint64_t)idx, DICT_BW);
            bw.putN(right, D.rightBW);
        }else{
            bw.putN(0, DICT_BW);
            bw.putN(right, D.rightBW);
            exc_pos[exc_cnt]  = (uint16_t)i;
            exc_left[exc_cnt] = left;
            ++exc_cnt;
        }
    }
    // 字典
    for(int k=0;k<DICT_SZ;++k) bw.putN((uint64_t)D.dict[k], D.leftBW);
    // 异常
    bw.putN((uint64_t)exc_cnt, 16);
    for(int i=0;i<exc_cnt;++i){
        bw.putN((uint64_t)exc_left[i], D.leftBW);
        bw.putN((uint64_t)exc_pos[i], 16);
    }
}

// ============= 每块：统计位数（含行组头 + 全向量） =============
template<typename T>
__device__ uint64_t measure_block_bits(const T* blk, int n, int vectorSize, CompressionMode& mode){
    int stride = (n/1024>0? n/1024 : 1);
    mode = decide_mode_block<T>(blk, n, stride);

    int numVec = (n + vectorSize - 1)/vectorSize;
    uint64_t bits = 8; // rowGroup.size()

    if (mode==CompressionMode::ALP){
        for(int v=0; v<numVec; ++v){
            int beg = v*vectorSize;
            int rem = n-beg;
            int len = (vectorSize < rem ? vectorSize : rem);
            uint8_t best_e=0, best_f=0; int best_exc=len+1; short bitw=0; long long FOR=0; int exc=0;
            for(uint8_t e=0;e<=18;++e){
                for(uint8_t f=0;f<=e;++f){
                    short _bitw; long long _FOR; int _exc;
                    alp_vector_analyze<T>(blk+beg, len, e, f, _bitw, _FOR, _exc);
                    if (_exc<best_exc){ best_exc=_exc; best_e=e; best_f=f; bitw=_bitw; FOR=_FOR; exc=_exc; }
                }
            }
            bits += alp_vector_size_bits<T>(len, best_e, best_f, bitw, exc);
        }
    }else{
        for(int v=0; v<numVec; ++v){
            int beg=v*vectorSize;
            int rem=n-beg;
            int len=(vectorSize<rem?vectorSize:rem);
            uint64_t tmp[MAX_VEC];
            assert(len <= MAX_VEC);
            for(int i=0;i<len;++i){
                if constexpr (std::is_same_v<T,double>) tmp[i] = *reinterpret_cast<const uint64_t*>(&blk[beg+i]);
                else tmp[i] = *reinterpret_cast<const uint32_t*>(&blk[beg+i]);
            }
            ALPrdDict<T> D; alprd_find_best<T>(tmp, len, D);

            // 统计异常（用小映射）
            int exc=0;
            for(int i=0;i<len;++i){
                uint32_t left = (uint32_t)((tmp[i] >> D.rightBW) & mask_lo(D.leftBW));
                bool inDict=false; for(int k=0;k<DICT_SZ;++k) if (D.dict[k]==left){ inDict=true; break; }
                if (!inDict) ++exc;
            }
            bits += alprd_vector_size_bits<T>(len, D, exc);
        }
    }
    return bits;
}

// ============= 每块：实际写入（按 offset） =============
template<typename T>
__device__ void emit_block_bits(const T* blk, int n, int vectorSize,
                                CompressionMode mode, BitWriter& bw){
    int numVec = (n + vectorSize - 1)/vectorSize;
    bw.putN((uint64_t)numVec, 8); // 行组头

    if (mode==CompressionMode::ALP){
        for(int v=0; v<numVec; ++v){
            int beg=v*vectorSize;
            int rem=n-beg;
            int len=(vectorSize<rem?vectorSize:rem);
            uint8_t best_e=0, best_f=0; int best_exc=len+1; short bitw=0; long long FOR=0; int exc=0;
            for(uint8_t e=0;e<=18;++e){
                for(uint8_t f=0;f<=e;++f){
                    short _bitw; long long _FOR; int _exc;
                    alp_vector_analyze<T>(blk+beg, len, e, f, _bitw, _FOR, _exc);
                    if (_exc<best_exc){ best_exc=_exc; best_e=e; best_f=f; bitw=_bitw; FOR=_FOR; exc=_exc; }
                }
            }
            alp_vector_write<T>(bw, blk+beg, len, best_e, best_f, bitw, FOR);
        }
    } else {
        for(int v=0; v<numVec; ++v){
            int beg=v*vectorSize;
            int rem=n-beg;
            int len=(vectorSize<rem?vectorSize:rem);
            uint64_t tmp[MAX_VEC];
            assert(len <= MAX_VEC);
            for(int i=0;i<len;++i){
                if constexpr (std::is_same_v<T,double>) tmp[i] = *reinterpret_cast<const uint64_t*>(&blk[beg+i]);
                else tmp[i] = *reinterpret_cast<const uint32_t*>(&blk[beg+i]);
            }
            ALPrdDict<T> D; alprd_find_best<T>(tmp, len, D);
            alprd_vector_write<T>(bw, tmp, len, D);
        }
    }
}

// ===================== Kernels：size & compress & decompress =====================
template<typename T>
__global__ void kernel_size_and_mode(const T* data, const uint64_t* blk_starts,
                                     const uint64_t* blk_sizes, int numBlocks,
                                     int vectorSize,
                                     uint64_t* out_bits, uint8_t* out_mode){
    int i = blockIdx.x; // 一线程一块
    if (i>=numBlocks) return;
    const T* blk = data + blk_starts[i];
    int n = (int)blk_sizes[i];
    CompressionMode mode;
    uint64_t bits = measure_block_bits<T>(blk, n, vectorSize, mode);
    out_bits[i] = bits;
    out_mode[i] = (mode==CompressionMode::ALPrd)?1:0;
}

template<typename T>
__global__ void kernel_compress_emit(const T* data,
                                     const uint64_t* blk_starts,
                                     const uint64_t* blk_sizes,
                                     const uint64_t* bit_offsets,
                                     const uint8_t*  modes,
                                     int numBlocks, int vectorSize,
                                     uint8_t* out_bytes){
    int i = blockIdx.x; // 一线程一块
    if (i>=numBlocks) return;
    const T* blk = data + blk_starts[i];
    int n = (int)blk_sizes[i];
    BitWriter bw{out_bytes, bit_offsets[i]};
    CompressionMode mode = (modes[i]?CompressionMode::ALPrd:CompressionMode::ALP);
    emit_block_bits<T>(blk, n, vectorSize, mode, bw);
}

template<typename T>
__global__ void kernel_decompress(const uint8_t* bytes,
                                  const uint64_t* blk_starts_bits,
                                  const uint64_t* /*blk_bits*/,
                                  const uint64_t* out_starts,
                                  const int vectorSize,
                                  T* out_data, int numBlocks){
    int i = blockIdx.x; // 一线程一块
    if (i>=numBlocks) return;

    BitReader br{bytes, blk_starts_bits[i]};
    int numVec = (int)br.getN(8);

    uint64_t out_pos = out_starts[i];
    for(int v=0; v<numVec; ++v){
        int useALP = br.get1();
        if (useALP){
            uint8_t e = (uint8_t)br.getN(8);
            uint8_t f = (uint8_t)br.getN(8);
            short bitw = (short)br.getN(16);
            long long FOR = (long long)br.getN(64);
            int n = (int)br.getN(32);
            assert(n <= MAX_VEC);
            for(int k=0;k<n;++k){
                uint64_t enc = br.getN(bitw);
                long long I = FOR + (long long)enc;
                double dec = double(I) * (1.0/double(D_FRAC_ARR[f])) * D_FRAC_ARR[e];
                out_data[out_pos + k] = (T)dec;
            }
            int exc = (int)br.getN(16);
            for(int t=0;t<exc;++t){
                uint64_t raw = std::is_same_v<T,double> ? br.getN(64) : br.getN(32);
                int pos = (int)br.getN(16);
                if constexpr (std::is_same_v<T,double>){
                    double val = *reinterpret_cast<double*>(&raw);
                    out_data[out_pos + pos] = (T)val;
                } else {
                    uint32_t rv = (uint32_t)raw;
                    float val = *reinterpret_cast<float*>(&rv);
                    out_data[out_pos + pos] = (T)val;
                }
            }
            out_pos += n;
        }else{
            int n = (int)br.getN(32);
            assert(n <= MAX_VEC);
            uint8_t rbw = (uint8_t)br.getN(8);
            uint64_t right[MAX_VEC]; uint16_t leftIdx[MAX_VEC];
            for(int k=0;k<n;++k){
                leftIdx[k] = (uint16_t)br.getN(DICT_BW);
                right[k]   = br.getN(rbw);
            }
            uint8_t lbw = uint8_t(sizeof(T)*8 - rbw);
            uint64_t dict[DICT_SZ];
            for(int k=0;k<DICT_SZ;++k) dict[k] = br.getN(lbw);

            int exc = (int)br.getN(16);
            uint16_t exc_pos[MAX_VEC]; uint64_t exc_left[MAX_VEC];
            for(int t=0;t<exc;++t){
                exc_left[t] = br.getN(lbw);
                exc_pos[t]  = (uint16_t)br.getN(16);
            }
            for(int k=0;k<n;++k){
                uint64_t left = (leftIdx[k]<DICT_SZ)? dict[leftIdx[k]] : 0ULL;
                uint64_t raw  = (left<<rbw) | right[k];
                if constexpr (std::is_same_v<T,double>){
                    double val = *reinterpret_cast<double*>(&raw);
                    out_data[out_pos + k] = (T)val;
                }else{
                    uint32_t r32 = (uint32_t)raw;
                    float val = *reinterpret_cast<float*>(&r32);
                    out_data[out_pos + k] = (T)val;
                }
            }
            for(int t=0;t<exc;++t){
                int p = exc_pos[t];
                uint64_t raw = (exc_left[t]<<rbw) | right[p];
                if constexpr (std::is_same_v<T,double>){
                    double val = *reinterpret_cast<double*>(&raw);
                    out_data[out_pos + p] = (T)val;
                }else{
                    uint32_t r32 = (uint32_t)raw;
                    float val = *reinterpret_cast<float*>(&r32);
                    out_data[out_pos + p] = (T)val;
                }
            }
            out_pos += n;
        }
    }
}

// ===================== Host 侧封装 =====================
template<typename T>
static Compressed compress_impl(const T* h_data, size_t n, const Params& p){
    Compressed c;
    if (n==0) { c.vectorSize = p.vectorSize; return c; }

    // 切块：一个线程处理一个数据块
    const int V = p.vectorSize;
    const int B = p.blockSize>0 ? p.blockSize : int(n);
    const int numBlocks = int( (n + B - 1)/B );

    std::vector<uint64_t> h_starts(numBlocks), h_sizes(numBlocks);
    size_t pos=0;
    for(int i=0;i<numBlocks;++i){
        h_starts[i]=pos;
        uint64_t sz = std::min<uint64_t>(B, n-pos);
        h_sizes[i]  = sz;
        pos += sz;
    }

    // 上传输入与分块表
    T* d_data=nullptr; hipMalloc(&d_data, n*sizeof(T));
    hipMemcpy(d_data, h_data, n*sizeof(T), hipMemcpyHostToDevice);

    uint64_t *d_starts=nullptr, *d_sizes=nullptr;
    hipMalloc(&d_starts, numBlocks*sizeof(uint64_t));
    hipMalloc(&d_sizes,  numBlocks*sizeof(uint64_t));
    hipMemcpy(d_starts, h_starts.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_sizes,  h_sizes.data(),  numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);

    // 第一阶段：测位数 & 模式（每块 1 线程）
    uint64_t* d_bits=nullptr;  uint8_t* d_mode=nullptr;
    hipMalloc(&d_bits, numBlocks*sizeof(uint64_t));
    hipMalloc(&d_mode, numBlocks*sizeof(uint8_t));

    dim3 gs(numBlocks), bs(1);
    kernel_size_and_mode<T><<<gs,bs>>>(d_data, d_starts, d_sizes, numBlocks, V, d_bits, d_mode);
    hipDeviceSynchronize();

    std::vector<uint64_t> h_bits(numBlocks); std::vector<uint8_t> h_mode(numBlocks);
    hipMemcpy(h_bits.data(), d_bits, numBlocks*sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_mode.data(), d_mode, numBlocks*sizeof(uint8_t),   hipMemcpyDeviceToHost);

    // exclusive-scan 计算 bit 偏移
    // std::vector<uint64_t> h_off(numBlocks);
    // uint64_t acc=0; for(int i=0;i<numBlocks;++i){ h_off[i]=acc; acc+=h_bits[i]; }
    // const uint64_t total_bits  = acc;


    // --- change ---
    std::vector<uint64_t> h_off(numBlocks), padded_bits(numBlocks);
    uint64_t acc = 0;
    for (int i = 0; i < numBlocks; ++i) {
        h_off[i] = acc;
        uint64_t bits = h_bits[i];
        // 对除了最后一块之外，都向上补齐到 32bit 边界
        uint64_t pad = (i + 1 < numBlocks) ? ((32 - (bits & 31ULL)) & 31ULL) : 0ULL;
        padded_bits[i] = bits + pad;
        acc += padded_bits[i];
    }
    const uint64_t total_bits  = acc;
    const uint64_t total_bytes = (total_bits + 7) / 8;



    // 分配输出 buffer
    uint8_t* d_out=nullptr; hipMalloc(&d_out, total_bytes);
    hipMemset(d_out, 0, total_bytes);
    uint64_t* d_off=nullptr; hipMalloc(&d_off, numBlocks*sizeof(uint64_t));
    hipMemcpy(d_off, h_off.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);

    // 第二阶段：真正写入
    kernel_compress_emit<T><<<gs,bs>>>(d_data, d_starts, d_sizes, d_off, d_mode, numBlocks, V, d_out);
    hipDeviceSynchronize();

    // 拷回结果 & per-block 元信息
    c.data.resize(total_bytes);
    hipMemcpy(c.data.data(), d_out, total_bytes, hipMemcpyDeviceToHost);
    c.offsets = std::move(h_off);
    c.bit_sizes = std::move(h_bits);
    c.elem_counts.assign(h_sizes.begin(), h_sizes.end());
    c.vectorSize = V;

    // 清理
    hipFree(d_out); hipFree(d_off);
    hipFree(d_mode); hipFree(d_bits);
    hipFree(d_sizes); hipFree(d_starts);
    hipFree(d_data);

    return c;
}

template<typename T>
static void decompress_impl(const Compressed& c, T* h_out, size_t n, const Params& p){
    if (n==0) return;
    const int numBlocks = (int)c.offsets.size();
    assert((size_t)numBlocks == c.elem_counts.size());

    // 上传压缩位流与 per-block 信息
    uint8_t* d_bytes=nullptr; hipMalloc(&d_bytes, c.data.size());
    hipMemcpy(d_bytes, c.data.data(), c.data.size(), hipMemcpyHostToDevice);

    uint64_t *d_boff=nullptr, *d_bsiz=nullptr, *d_ost=nullptr;
    hipMalloc(&d_boff, numBlocks*sizeof(uint64_t));
    hipMalloc(&d_bsiz, numBlocks*sizeof(uint64_t));
    hipMemcpy(d_boff, c.offsets.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_bsiz, c.bit_sizes.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);

    // 由 elem_counts 前缀和得到每块输出起点
    std::vector<uint64_t> h_outStarts(numBlocks);
    uint64_t acc=0; for(int i=0;i<numBlocks;++i){ h_outStarts[i]=acc; acc+=c.elem_counts[i]; }
    assert(acc == n && "elem_counts 总和必须等于输出元素数");

    hipMalloc(&d_ost, numBlocks*sizeof(uint64_t));
    hipMemcpy(d_ost, h_outStarts.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);

    T* d_out=nullptr; hipMalloc(&d_out, n*sizeof(T));

    dim3 gs(numBlocks), bs(1);
    kernel_decompress<T><<<gs,bs>>>(d_bytes, d_boff, d_bsiz, d_ost, p.vectorSize, d_out, numBlocks);
    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, n*sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_ost);
    hipFree(d_bsiz);
    hipFree(d_boff);
    hipFree(d_bytes);
}

// 显式实例化 API
Compressed compress_double(const double* data, size_t n, const Params& p){ return compress_impl<double>(data,n,p); }
Compressed compress_float (const float*  data, size_t n, const Params& p){ return compress_impl<float >(data,n,p); }
void decompress_double(const Compressed& c, double* out, size_t n, const Params& p){ decompress_impl<double>(c,out,n,p); }
void decompress_float (const Compressed& c, float*  out, size_t n, const Params& p){ decompress_impl<float >(c,out,n,p); }

} // namespace alp_gpu