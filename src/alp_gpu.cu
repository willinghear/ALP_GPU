#include "hip/hip_runtime.h"
/*
 * ============================================
 *  ALP-GPU 压缩/解压（整理版）
 *  设计要点：
 *    1) 全局量化预测（ALP 模式）：对整段数据尝试 e/f 组合，异常值单独存储；
 *    2) 浮点切割（ALPrd 模式）：bit 切割 + 左半部分字典 + 右半部分直写；
 *    3) 数据划分：将原始数组切成多个“数据块”，每个线程处理一个块；
 *    4) 写出格式：位粒度写入，记录每块起始 offset(bit) 与占用 bit 数；
 *    5) CUDA 并行：网格=块数/threadsPerBlock，blockDim=threadsPerBlock；
 *  与 CPU 版的差异：
 *    - CPU 逐向量/行组串行处理；GPU 以“数据块”为单位并行，线程内再做向量切分；
 *    - GPU 使用位写流(BitWriter)与原子加总做块内统计；
 *  本文件仅整理注释与移除无意义代码，算法逻辑保持不变以确保结果对齐。
 * ============================================
 */

/*该版本代码的选择器与CPU 版本一致，尽可能的提高了压缩率 */
#include "alp_gpu.hpp"
#include <hip/hip_runtime.h>
#include <cassert>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <stdexcept>
#include <iostream>
#include <climits> // LLONG_MAX, LLONG_MIN
#include <cstdlib> // getenv
#include <vector>

using std::uint8_t; using std::uint32_t; using std::uint64_t;

namespace alp_gpu {

// ===================== 两级采样配置（与CPU版本对齐） =====================
namespace sampling_config {
    static constexpr int ROWGROUP_SIZE = 100000;          // 行组大小（与blockSize一致）
    static constexpr int ROWGROUP_VECTOR_SAMPLES = 8;     // 行组级采样向量数
    static constexpr int SAMPLES_PER_VECTOR = 32;         // 每向量采样数
    static constexpr int MAX_K_COMBINATIONS = 5;          // 最多保留k个组合
    static constexpr int EARLY_EXIT_THRESHOLD = 2;        // 早退阈值
}

// 阈值定义（与CPU版本一致）
template<typename T> struct SamplingConstants {
    // RD切换阈值：每采样值的平均位数
    static constexpr size_t RD_SIZE_THRESHOLD_LIMIT = 
        sizeof(T) == 8 ? (48 * sampling_config::SAMPLES_PER_VECTOR) 
                       : (22 * sampling_config::SAMPLES_PER_VECTOR);
};


// ===================== 常量（与 CPU 版一致） =====================
// e,f 取值范围 0..18（19个）
__device__ __constant__ double D_EXP_ARR[19] = {
  1.0, 10.0, 100.0, 1000.0, 10000.0, 100000.0, 1000000.0, 10000000.0,
  100000000.0, 1000000000.0, 10000000000.0, 100000000000.0, 1000000000000.0,
  10000000000000.0, 100000000000000.0, 1000000000000000.0,
  10000000000000000.0, 100000000000000000.0, 1000000000000000000.0
};
__device__ __constant__ double D_FRAC_ARR[20] = {
  1.0, 0.1, 0.01, 0.001, 0.0001, 0.00001, 0.000001, 0.0000001,
  0.00000001, 0.000000001, 0.0000000001, 0.00000000001, 0.000000000001,
  0.0000000000001, 0.00000000000001, 0.000000000000001, 0.0000000000000001,
  0.00000000000000001, 0.000000000000000001
};
// ALPrd：3bit 字典（大小 8），CUTTING_LIMIT = 16（与 CPU 保持）
static constexpr int   DICT_BW  = 3;
static constexpr int   DICT_SZ  = 1 << DICT_BW;
static constexpr int   CUT_LIM  = 16;
static constexpr int   MAX_VEC  = 4096; // 保护上限（<= 4096）

// ===================== 设备端位流 Writer/Reader =====================
struct BitWriter {
    uint8_t* buf;      // 全局输出缓冲（按 bit 偏移写）
    uint64_t bitpos;   // 写入起始 bit 偏移（每块各自独立）
    __device__ void put1(int b){
        if (!b) { ++bitpos; return; }          // 初始缓冲区已清零
        uint64_t byte = bitpos >> 3;
        int      off  = 7 - int(bitpos & 7ULL);
        buf[byte] = uint8_t(buf[byte] | (uint8_t(1u) << off));  // 普通字节写
        ++bitpos;
    }
    __device__ void putN(uint64_t v, int bits){
        for(int i=bits-1;i>=0;--i) put1( (v>>i) & 1ULL );
    }
};
struct BitReader {
    const uint8_t* buf;
    uint64_t bitpos;
    __device__ int get1(){
        uint64_t byte = bitpos >> 3;
        int off = 7 - int(bitpos & 7ULL);
        int b = (buf[byte] >> off) & 1;
        ++bitpos; return b;
    }
    __device__ uint64_t getN(int bits){
        uint64_t v=0;
        for(int i=0;i<bits;++i){ v = (v<<1) | get1(); }
        return v;
    }
};

// ---- MSB-first bit reader (device) ----
struct DevBitReader {
    const unsigned char* base;  // 指向整个压缩字节流
    unsigned long long bitpos;  // 全局位偏移（以bit为单位）

    __device__ DevBitReader(const unsigned char* p, unsigned long long start_bits)
        : base(p), bitpos(start_bits) {}

    __device__ inline unsigned int get1() {
        unsigned long long byte_idx = bitpos >> 3;
        int inbyte = (int)(bitpos & 7ULL);
        unsigned char b = base[byte_idx];
        // MSB-first：bit顺序为 7,6,...,0
        int shift = 7 - inbyte;
        unsigned int v = (b >> shift) & 1u;
        ++bitpos;
        return v;
    }
    __device__ inline unsigned long long getN(int n) {
        unsigned long long v = 0ULL;
        #pragma unroll
        for (int i = 0; i < n; ++i) {
            v = (v << 1) | (unsigned long long)get1();
        }
        return v;
    }
    __device__ inline unsigned long long pos() const { return bitpos; }
};


// ===================== 公共工具（GPU） =====================
__device__ __forceinline__ int width_needed_unsigned(unsigned long long range){
    if (range==0ULL) return 1;
    int c=0; while(range){ ++c; range>>=1ULL; } return c;
}

__device__ inline long long fast_round_double(double x){
    // 与 CPU 版相同的“甜 spot”整型回转
    const double SWEET = double((1ULL<<51) + (1ULL<<52));
    return (long long)(x + SWEET) - (long long)SWEET;
}

// ============= 采样 & 模式判定（与 CPU 逻辑对齐） =============
template<typename T>
__device__ inline bool alp_exact_equal(T v, uint8_t e, uint8_t f){
    if constexpr (std::is_same_v<T,double>) {
        double enc = v * D_EXP_ARR[e] * D_FRAC_ARR[f];
        long long I = fast_round_double(enc);
        double dec = double(I) * (1.0 / D_FRAC_ARR[f]) * D_FRAC_ARR[e];
        return dec==v;
    } else {
        float enc = v * float(D_EXP_ARR[e]) * float(D_FRAC_ARR[f]);
        int   I   = __float2int_rn(enc);
        float dec = float(I) * (1.0f/float(D_FRAC_ARR[f])) * float(D_FRAC_ARR[e]);
        return dec==v;
    }
}

template<typename T>
__device__ inline bool is_high_precision_value(T v){
    for(uint8_t e=0;e<=18;++e){
        for(uint8_t f=0; f<=e; ++f){
            if (alp_exact_equal<T>(v,e,f)) return false;
        }
    }
    return true;
}

// ============= ALP 单向量：统计 & 写入 =============
template<typename T>
__device__ inline void alp_vector_analyze(const T* v, int n, uint8_t e, uint8_t f,
                                          short& bitw, long long& FOR,
                                          int& exc_cnt){
    long long mn=LLONG_MAX, mx=LLONG_MIN;
    exc_cnt=0;
    for(int i=0;i<n;++i){
        double enc = double(v[i]) * D_EXP_ARR[e] * D_FRAC_ARR[f];
        long long I = fast_round_double(enc);
        double dec = double(I) * (1.0/double(D_FRAC_ARR[f])) * D_FRAC_ARR[e];
        if (dec==double(v[i])) { mn=(mn<I?mn:I); mx=(mx>I?mx:I); }
        else ++exc_cnt;
    }
    unsigned long long range = (mn==LLONG_MAX)? 0ULL : (unsigned long long)(mx - mn);
    bitw = (short)width_needed_unsigned(range);
    FOR  = (mn==LLONG_MAX?0:mn);
}

// 选择使“估计总比特”最小的 (e,f)
// 返回同时带回 bitw/FOR/异常数，便于后续直接写流。
template<typename T>
__device__ inline void alp_vector_choose_best_bits(
    const T* v, int n,
    uint8_t& best_e, uint8_t& best_f,
    short& bitw, long long& FOR, int& exc)
{
    const int val_bits = std::is_same_v<T,double> ? 64 : 32;
    double best_score = 1e300;
    best_e=0; best_f=0; bitw=0; FOR=0; exc=0;

    for(uint8_t e=0;e<=18;++e){
        for(uint8_t f=0;f<=e;++f){
            short _bw; long long _FOR; int _exc;
            alp_vector_analyze<T>(v, n, e, f, _bw, _FOR, _exc);
            // 估计：打包体(n*_bw) + 异常表(_exc*(值位数+位置16位)) + 头部开销(常数)，
            // 比较时可以忽略常数项（对所有(e,f)相同），保留可变部分更快。
            double score = double(n)*_bw + double(_exc)*(val_bits + 16);
            if (score < best_score){
                best_score = score;
                best_e = e; best_f = f; bitw = _bw; FOR = _FOR; exc = _exc;
            }
        }
    }
}


template<typename T>
__device__ inline uint64_t alp_vector_size_bits(int n, uint8_t e, uint8_t f,
                                                short bitw, int exc_cnt){
    int val_bits = std::is_same_v<T,double> ? 64 : 32;
    return 1 + 8+8+16+64+32 + uint64_t(n)*bitw + 16 + uint64_t(exc_cnt)*(val_bits+16);
}

template<typename T>
__device__ inline void alp_vector_write(BitWriter& bw, const T* v, int n,
                                       uint8_t e, uint8_t f, short bitw, long long FOR){
    assert(n <= MAX_VEC);
    bw.put1(1); // useALP = 1
    bw.putN((uint64_t)e, 8); bw.putN((uint64_t)f, 8);
    bw.putN((uint64_t)bitw, 16);
    bw.putN((uint64_t)FOR, 64);
    bw.putN((uint64_t)n, 32);

    int exc_cnt=0;
    int      exc_pos[MAX_VEC];
    uint64_t exc_val[MAX_VEC];

    for(int i=0;i<n;++i){
        double enc = double(v[i]) * D_EXP_ARR[e] * D_FRAC_ARR[f];
        long long I = fast_round_double(enc);
        double dec = double(I) * (1.0/double(D_FRAC_ARR[f])) * D_FRAC_ARR[e];
        if (dec==double(v[i])) {
            uint64_t packed = (uint64_t)(I - FOR);
            bw.putN(packed, bitw);
        } else {
            bw.putN(0, bitw); // 占位
            if constexpr (std::is_same_v<T,double>) {
                uint64_t raw = *reinterpret_cast<const uint64_t*>(&v[i]);
                exc_val[exc_cnt] = raw;
            } else {
                uint32_t raw = *reinterpret_cast<const uint32_t*>(&v[i]);
                exc_val[exc_cnt] = raw;
            }
            exc_pos[exc_cnt] = i;
            ++exc_cnt;
        }
    }
    bw.putN((uint64_t)exc_cnt, 16);
    for(int k=0;k<exc_cnt;++k){
        if constexpr (std::is_same_v<T,double>) bw.putN(exc_val[k], 64);
        else bw.putN(exc_val[k], 32);
        bw.putN((uint64_t)exc_pos[k], 16);
    }
}

// ============= ALPrd：字典与写入（精确 top-8，本地小表，无 shared） =============
template<typename T> struct ALPrdDict {
    uint8_t rightBW;        // 右半位宽
    uint8_t leftBW;         // 左半位宽
    uint32_t dict[DICT_SZ]; // left parts 字典（<= 2^leftBW-1）
};

__device__ inline uint32_t mask_lo(int bits){
    return (bits >= 32) ? 0xFFFFFFFFu : ((1u<<bits) - 1u);
}

template<typename T>
__device__ inline void alprd_find_best(const uint64_t* in, int n, ALPrdDict<T>& D){
    // 穷举左宽 1..CUT_LIM，精确统计频次 → 取 top-8
    double best_score = 1e100; int best_rbw = int(sizeof(T)*8) - 1;
    uint32_t best_dict[DICT_SZ] = {0};

    for(int lbw=1; lbw<=CUT_LIM; ++lbw){
        int rbw = int(sizeof(T)*8) - lbw;
        uint32_t lmask = mask_lo(lbw);

        // 频次统计（最多 n=vectorSize 个不同 left），用小表（最多 n 项）
        uint32_t uniq_left[MAX_VEC]; int cnt[MAX_VEC];
        int u = 0;

        for(int i=0;i<n;++i){
            uint32_t left = (uint32_t)((in[i] >> rbw) & lmask);
            int j=0; for(; j<u; ++j) if (uniq_left[j]==left) { ++cnt[j]; break; }
            if (j==u){ uniq_left[u]=left; cnt[u]=1; ++u; }
        }
        // 选 top-8
        uint32_t dict[DICT_SZ]={0};
        int used = (DICT_SZ < u ? DICT_SZ : u);
        for(int k=0;k<used;++k){
            int best=-1, id=-1;
            for(int j=0;j<u;++j){
                bool taken=false;
                for(int t=0;t<k;++t) if (dict[t]==uniq_left[j]) { taken=true; break; }
                if (taken) continue;
                if (cnt[j]>best){ best=cnt[j]; id=j; }
            }
            dict[k] = uniq_left[id];
        }
        // 异常数量
        int keep=0;
        for(int k=0;k<used;++k){
            for(int j=0;j<u;++j) if (uniq_left[j]==dict[k]) { keep += cnt[j]; break; }
        }
        int exc = n - keep;

        // 估计位数：n*(DICT_BW+rbw) + dict(8*lbw) + 异常(16+lbw)*exc + 头(1+32+8)
        double bits = 1 + 32 + 8
                      + double(n)*(DICT_BW + rbw)
                      + double(DICT_SZ)*lbw
                      + 16.0*exc + double(lbw)*exc;

        if (bits < best_score){
            best_score = bits;
            best_rbw   = rbw;
            for(int k=0;k<DICT_SZ;++k) best_dict[k]=dict[k];
        }
    }
    D.rightBW = (uint8_t)best_rbw;
    D.leftBW  = (uint8_t)(int(sizeof(T)*8) - best_rbw);
    for(int k=0;k<DICT_SZ;++k) D.dict[k]=best_dict[k];
}

template<typename T>
__device__ inline uint64_t alprd_vector_size_bits(int n, const ALPrdDict<T>& D, int exc_cnt){
    return 1 + 32 + 8 + uint64_t(n)*(DICT_BW + D.rightBW)
           + DICT_SZ*D.leftBW + 16 + uint64_t(exc_cnt)*(D.leftBW+16);
}

template<typename T>
__device__ inline void alprd_vector_write(BitWriter& bw, const uint64_t* in, int n,
                                          const ALPrdDict<T>& D){
    assert(n <= MAX_VEC);
    bw.put1(0); // useALP=0
    bw.putN((uint64_t)n, 32);
    bw.putN((uint64_t)D.rightBW, 8);

    // 逐值输出：leftIdx(3)+right(rbw)，同时记录异常
    int exc_cnt=0; uint16_t exc_pos[MAX_VEC]; uint32_t exc_left[MAX_VEC];
    uint64_t right_mask = (D.rightBW==64)? ~0ULL : ((1ULL<<D.rightBW)-1ULL);
    uint32_t left_mask  = mask_lo(D.leftBW);

    for(int i=0;i<n;++i){
        uint64_t right = in[i] & right_mask;
        uint32_t left  = (uint32_t)((in[i] >> D.rightBW) & left_mask);
        short idx = DICT_SZ;
        for(int k=0;k<DICT_SZ;++k){ if (D.dict[k]==left){ idx=(short)k; break; } }
        if (idx<DICT_SZ){
            bw.putN((uint64_t)idx, DICT_BW);
            bw.putN(right, D.rightBW);
        }else{
            bw.putN(0, DICT_BW);
            bw.putN(right, D.rightBW);
            exc_pos[exc_cnt]  = (uint16_t)i;
            exc_left[exc_cnt] = left;
            ++exc_cnt;
        }
    }
    // 字典
    for(int k=0;k<DICT_SZ;++k) bw.putN((uint64_t)D.dict[k], D.leftBW);
    // 异常
    bw.putN((uint64_t)exc_cnt, 16);
    for(int i=0;i<exc_cnt;++i){
        bw.putN((uint64_t)exc_left[i], D.leftBW);
        bw.putN((uint64_t)exc_pos[i], 16);
    }
}

// ===================== 第一级采样：行组级 =====================
template<typename T>
__device__ void rowgroup_sample_and_find_k_combinations(
    const T* rowgroup_data, 
    int rowgroup_size,
    int vectorSize,
    EFCombination* best_k_combinations,  // 输出：k个最佳组合
    int& k_actual,                       // 输出：实际找到的组合数
    CompressionMode& mode                // 输出：ALP或ALPrd
) {
    // 计算采样步长
    int total_vectors = (rowgroup_size + vectorSize - 1) / vectorSize;
    int sample_stride = max(1, total_vectors / sampling_config::ROWGROUP_VECTOR_SAMPLES);
    
    // 统计每个(e,f)组合的表现
    struct LocalStats {
        int count;
        double total_score;
    } stats[19][19];  // 最多19x19种组合
    
    // 初始化
    for(int e=0; e<=18; e++) {
        for(int f=0; f<=e; f++) {
            stats[e][f].count = 0;
            stats[e][f].total_score = 0;
        }
    }
    
    double best_overall_compression_size = 1e30;
    int samples_taken = 0;
    
    // 采样向量
    for(int v = 0; v < total_vectors && samples_taken < sampling_config::ROWGROUP_VECTOR_SAMPLES; 
        v += sample_stride) {
        
        int vec_start = v * vectorSize;
        int vec_size = min(vectorSize, rowgroup_size - vec_start);
        if(vec_size <= 0) break;
        
        // 从该向量采样
        T samples[32];  // SAMPLES_PER_VECTOR
        int sample_count = min(sampling_config::SAMPLES_PER_VECTOR, vec_size);
        int sample_step = max(1, vec_size / sample_count);
        
        for(int i = 0; i < sample_count; i++) {
            samples[i] = rowgroup_data[vec_start + i * sample_step];
        }
        
        // 找该采样向量的最佳(e,f)
        uint8_t best_e = 0, best_f = 0;
        short bitw; long long FOR; int exc;
        alp_vector_choose_best_bits<T>(samples, sample_count, best_e, best_f, bitw, FOR, exc);
        
        // 计算压缩大小评分
        int val_bits = std::is_same_v<T,double> ? 64 : 32;
        double compression_size = sample_count * bitw + exc * (val_bits + 16);
        
        // 记录该向量选择的(e,f)
        stats[best_e][best_f].count++;
        stats[best_e][best_f].total_score += compression_size;
        
        if(compression_size < best_overall_compression_size) {
            best_overall_compression_size = compression_size;
        }
        
        samples_taken++;
    }
    
    // 判断是否切换到ALPrd模式（基于阈值）
    if(best_overall_compression_size >= SamplingConstants<T>::RD_SIZE_THRESHOLD_LIMIT) {
        mode = CompressionMode::ALPrd;
        k_actual = 0;
        return;
    }
    
    mode = CompressionMode::ALP;
    
    // 收集所有出现过的(e,f)组合
    EFCombination all_combinations[361];  
    int num_combinations = 0;
    
    for(int e = 0; e <= 18; e++) {
        for(int f = 0; f <= e; f++) {
            if(stats[e][f].count > 0) {
                all_combinations[num_combinations].e = e;
                all_combinations[num_combinations].f = f;
                all_combinations[num_combinations].count = stats[e][f].count;
                all_combinations[num_combinations].score = 
                    stats[e][f].total_score / stats[e][f].count;
                num_combinations++;
            }
        }
    }
    
    // 简单排序（冒泡排序，因为数量少）
    for(int i = 0; i < num_combinations - 1; i++) {
        for(int j = i + 1; j < num_combinations; j++) {
            bool swap = false;
            if(all_combinations[j].count > all_combinations[i].count) {
                swap = true;
            } else if(all_combinations[j].count == all_combinations[i].count) {
                if(all_combinations[j].score < all_combinations[i].score) {
                    swap = true;
                }
            }
            
            if(swap) {
                EFCombination tmp = all_combinations[i];
                all_combinations[i] = all_combinations[j];
                all_combinations[j] = tmp;
            }
        }
    }
    
    // 取前k个
    k_actual = min(sampling_config::MAX_K_COMBINATIONS, num_combinations);
    for(int i = 0; i < k_actual; i++) {
        best_k_combinations[i] = all_combinations[i];
    }
}

// ===================== 第二级采样：向量级 =====================
template<typename T>
__device__ void vector_choose_from_k_combinations(
    const T* vec_data,
    int vec_size,
    const EFCombination* k_combinations,
    int k,
    uint8_t& best_e,
    uint8_t& best_f,
    short& bitw,
    long long& FOR,
    int& exc
) {
    // 如果只有一个组合，直接使用
    if(k == 1) {
        best_e = k_combinations[0].e;
        best_f = k_combinations[0].f;
        alp_vector_analyze<T>(vec_data, vec_size, best_e, best_f, bitw, FOR, exc);
        return;
    }
    
    // 采样向量数据
    T samples[32];
    int sample_count = min(sampling_config::SAMPLES_PER_VECTOR, vec_size);
    int sample_step = max(1, vec_size / sample_count);
    
    for(int i = 0; i < sample_count; i++) {
        samples[i] = vec_data[i * sample_step];
    }
    
    // 在k个组合中选择最佳
    double best_score = 1e30;
    int worse_count = 0;
    
    for(int kid = 0; kid < k; kid++) {
        uint8_t e = k_combinations[kid].e;
        uint8_t f = k_combinations[kid].f;
        
        // 测试这个组合
        short test_bitw;
        long long test_FOR;
        int test_exc;
        alp_vector_analyze<T>(samples, sample_count, e, f, test_bitw, test_FOR, test_exc);
        
        int val_bits = std::is_same_v<T,double> ? 64 : 32;
        double score = sample_count * test_bitw + test_exc * (val_bits + 16);
        
        if(score < best_score) {
            best_score = score;
            best_e = e;
            best_f = f;
            worse_count = 0;
        } else {
            worse_count++;
            // 早退机制
            if(worse_count >= sampling_config::EARLY_EXIT_THRESHOLD) {
                break;
            }
        }
    }
    
    // 对完整向量应用选定的(e,f)
    alp_vector_analyze<T>(vec_data, vec_size, best_e, best_f, bitw, FOR, exc);
}

// ===================== 优化的Kernels：向量级并行 =====================


// 每个block的线程数（必须是32的倍数，warp大小）
static constexpr int THREADS_PER_BLOCK = 128;
static constexpr int MAX_VECS_PER_BLOCK = 256;  // 共享内存限制

template<typename T>
__global__ void kernel_decompress(const uint8_t* bytes,
                                  const uint64_t* blk_starts_bits,
                                  const uint64_t* /*blk_bits*/,
                                  const uint64_t* out_starts,
                                  const int vectorSize,
                                  T* out_data, int numBlocks){
    int i = blockIdx.x; // 一线程一块
    if (i>=numBlocks) return;

    BitReader br{bytes, blk_starts_bits[i]};
    int numVec = (int)br.getN(8);

    uint64_t out_pos = out_starts[i];
    for(int v=0; v<numVec; ++v){
        int useALP = br.get1();
        if (useALP){
            uint8_t e = (uint8_t)br.getN(8);
            uint8_t f = (uint8_t)br.getN(8);
            short bitw = (short)br.getN(16);
            long long FOR = (long long)br.getN(64);
            int n = (int)br.getN(32);
            assert(n <= MAX_VEC);
            for(int k=0;k<n;++k){
                uint64_t enc = br.getN(bitw);
                long long I = FOR + (long long)enc;
                double dec = double(I) * (1.0/double(D_FRAC_ARR[f])) * D_FRAC_ARR[e];
                out_data[out_pos + k] = (T)dec;
            }
            int exc = (int)br.getN(16);
            for(int t=0;t<exc;++t){
                uint64_t raw = std::is_same_v<T,double> ? br.getN(64) : br.getN(32);
                int pos = (int)br.getN(16);
                if constexpr (std::is_same_v<T,double>){
                    double val = *reinterpret_cast<double*>(&raw);
                    out_data[out_pos + pos] = (T)val;
                } else {
                    uint32_t rv = (uint32_t)raw;
                    float val = *reinterpret_cast<float*>(&rv);
                    out_data[out_pos + pos] = (T)val;
                }
            }
            out_pos += n;
        }else{
            int n = (int)br.getN(32);
            assert(n <= MAX_VEC);
            uint8_t rbw = (uint8_t)br.getN(8);
            uint64_t right[MAX_VEC]; uint16_t leftIdx[MAX_VEC];
            for(int k=0;k<n;++k){
                leftIdx[k] = (uint16_t)br.getN(DICT_BW);
                right[k]   = br.getN(rbw);
            }
            uint8_t lbw = uint8_t(sizeof(T)*8 - rbw);
            uint64_t dict[DICT_SZ];
            for(int k=0;k<DICT_SZ;++k) dict[k] = br.getN(lbw);

            int exc = (int)br.getN(16);
            uint16_t exc_pos[MAX_VEC]; uint64_t exc_left[MAX_VEC];
            for(int t=0;t<exc;++t){
                exc_left[t] = br.getN(lbw);
                exc_pos[t]  = (uint16_t)br.getN(16);
            }
            for(int k=0;k<n;++k){
                uint64_t left = (leftIdx[k]<DICT_SZ)? dict[leftIdx[k]] : 0ULL;
                uint64_t raw  = (left<<rbw) | right[k];
                if constexpr (std::is_same_v<T,double>){
                    double val = *reinterpret_cast<double*>(&raw);
                    out_data[out_pos + k] = (T)val;
                }else{
                    uint32_t r32 = (uint32_t)raw;
                    float val = *reinterpret_cast<float*>(&r32);
                    out_data[out_pos + k] = (T)val;
                }
            }
            for(int t=0;t<exc;++t){
                int p = exc_pos[t];
                uint64_t raw = (exc_left[t]<<rbw) | right[p];
                if constexpr (std::is_same_v<T,double>){
                    double val = *reinterpret_cast<double*>(&raw);
                    out_data[out_pos + p] = (T)val;
                }else{
                    uint32_t r32 = (uint32_t)raw;
                    float val = *reinterpret_cast<float*>(&r32);
                    out_data[out_pos + p] = (T)val;
                }
            }
            out_pos += n;
        }
    }
}

template<typename T>
__global__ void kernel_measure_with_sampling(
    const T* data, 
    const uint64_t* blk_starts,
    const uint64_t* blk_sizes, 
    int numBlocks,
    int vectorSize,
    uint64_t* out_bits,     
    uint8_t* out_mode      
) {
    int blockId = blockIdx.x;
    if (blockId >= numBlocks) return;
    
    const T* blk = data + blk_starts[blockId];
    int n = (int)blk_sizes[blockId];
    int numVec = (n + vectorSize - 1) / vectorSize;
    
    // 使用两级采样决定模式
    EFCombination k_combinations[5];
    int k_actual = 0;
    CompressionMode mode;
    
    // 第一级采样（线程0执行）
    if(threadIdx.x == 0) {
        rowgroup_sample_and_find_k_combinations<T>(
            blk, n, vectorSize,
            k_combinations, k_actual, mode
        );
    }
    __syncthreads();
    
    // 广播结果到所有线程
    __shared__ CompressionMode sh_mode;
    __shared__ EFCombination sh_k_combinations[5];
    __shared__ int sh_k_actual;
    
    if(threadIdx.x == 0) {
        sh_mode = mode;
        sh_k_actual = k_actual;
        for(int i = 0; i < k_actual; i++) {
            sh_k_combinations[i] = k_combinations[i];
        }
    }
    __syncthreads();
    
    // 计算总位数
    __shared__ uint64_t sh_total_bits;
    if(threadIdx.x == 0) {
        sh_total_bits = 8;  // 行组头
    }
    __syncthreads();
    
    if(sh_mode == CompressionMode::ALP) {
        // ALP模式：并行计算每个向量的位数
        for(int v = threadIdx.x; v < numVec; v += blockDim.x) {
            int beg = v * vectorSize;
            int rem = n - beg;
            int len = (vectorSize < rem ? vectorSize : rem);
            
            uint8_t e, f; short bw; long long FOR; int exc;
            
            // 使用第二级采样选择(e,f)
            vector_choose_from_k_combinations<T>(
                blk + beg, len,
                sh_k_combinations, sh_k_actual,
                e, f, bw, FOR, exc
            );
            
            uint64_t bits = alp_vector_size_bits<T>(len, e, f, bw, exc);
            atomicAdd((unsigned long long*)&sh_total_bits, bits);
        }
    } else {
        // ALPrd模式
        for(int v = threadIdx.x; v < numVec; v += blockDim.x) {
            int beg = v * vectorSize;
            int rem = n - beg;
            int len = (vectorSize < rem ? vectorSize : rem);
            
            uint64_t tmp[MAX_VEC];
            for(int i = 0; i < len; i++) {
                if constexpr (std::is_same_v<T,double>) 
                    tmp[i] = *reinterpret_cast<const uint64_t*>(&blk[beg+i]);
                else 
                    tmp[i] = *reinterpret_cast<const uint32_t*>(&blk[beg+i]);
            }
            
            ALPrdDict<T> D;
            alprd_find_best<T>(tmp, len, D);
            
            // 统计异常
            int exc = 0;
            for(int i = 0; i < len; i++) {
                uint32_t left = (uint32_t)((tmp[i] >> D.rightBW) & mask_lo(D.leftBW));
                bool inDict = false;
                for(int k = 0; k < DICT_SZ; k++) {
                    if(D.dict[k] == left) {
                        inDict = true;
                        break;
                    }
                }
                if(!inDict) exc++;
            }
            
            uint64_t bits = alprd_vector_size_bits<T>(len, D, exc);
            atomicAdd((unsigned long long*)&sh_total_bits, bits);
        }
    }
    
    __syncthreads();
    
    // 线程0写出结果
    if(threadIdx.x == 0) {
        out_bits[blockId] = sh_total_bits;
        out_mode[blockId] = (sh_mode == CompressionMode::ALPrd) ? 1 : 0;
    }
}
template<typename T>
__global__ void kernel_emit_with_sampling(
    const T* data,
    const uint64_t* blk_starts,
    const uint64_t* blk_sizes,
    const uint64_t* bit_offsets,
    const uint8_t* modes,
    const uint64_t* vec_prefix,
    uint8_t* dbg_modes, 
    uint8_t* dbg_e, 
    uint8_t* dbg_f,
    int enable_diag,
    int numBlocks, 
    int vectorSize,
    uint8_t* out_bytes
) {
    int blockId = blockIdx.x;
    if (blockId >= numBlocks) return;
    
    const T* blk = data + blk_starts[blockId];
    int n = (int)blk_sizes[blockId];
    int numVec = (n + vectorSize - 1) / vectorSize;
    CompressionMode mode = (modes[blockId] ? CompressionMode::ALPrd : CompressionMode::ALP);
    
    // 重新执行第一级采样以获取k个组合
    EFCombination k_combinations[5];
    int k_actual = 0;
    CompressionMode mode_check;
    if(threadIdx.x == 0) {
        rowgroup_sample_and_find_k_combinations<T>(
            blk, n, vectorSize,
            k_combinations, k_actual, mode_check
        );
    }
    __syncthreads();
    // 广播到共享内存
    __shared__ EFCombination sh_k_combinations[5];
    __shared__ int sh_k_actual;
    
    if(threadIdx.x == 0) {
        sh_k_actual = k_actual;
        for(int i = 0; i < k_actual; i++) {
            sh_k_combinations[i] = k_combinations[i];
        }
    }
    __syncthreads();

    // 限制：如果向量太多，使用单线程串行处理
    if (numVec > MAX_VECS_PER_BLOCK) {
        if (threadIdx.x == 0) {
            BitWriter bw{out_bytes, bit_offsets[blockId]};
            bw.putN((uint64_t)numVec, 8);  // 行组头
            
            if (mode == CompressionMode::ALP) {
                // ALP模式：对每个向量直接计算最优(e,f)
                for(int v = 0; v < numVec; v++) {
                    int beg = v * vectorSize;
                    int rem = n - beg;
                    int len = (vectorSize < rem ? vectorSize : rem);
                    
                    // 关键：基于完整向量数据选择最优(e,f)
                    // uint8_t e, f;
                    // short bitw;
                    // long long FOR;
                    // int exc;
                    // alp_vector_choose_best_bits<T>(blk + beg, len, e, f, bitw, FOR, exc);
                    uint8_t e, f; short bitw; long long FOR; int exc;
                
                    // 使用第二级采样
                    vector_choose_from_k_combinations<T>(
                        blk + beg, len,
                        sh_k_combinations, sh_k_actual,
                        e, f, bitw, FOR, exc
                    );

                    // 记录调试信息
                    if (enable_diag && dbg_modes) {
                        uint64_t gid = vec_prefix[blockId] + (uint64_t)v;
                        dbg_modes[gid] = 0;  // ALP
                        if (dbg_e) dbg_e[gid] = e;
                        if (dbg_f) dbg_f[gid] = f;
                    }
                    
                    // 写入压缩数据
                    alp_vector_write<T>(bw, blk + beg, len, e, f, bitw, FOR);
                }
            } else {
                // ALPrd模式
                for(int v = 0; v < numVec; v++) {
                    int beg = v * vectorSize;
                    int rem = n - beg;
                    int len = (vectorSize < rem ? vectorSize : rem);
                    
                    uint64_t tmp[MAX_VEC];
                    assert(len <= MAX_VEC);
                    for(int i = 0; i < len; i++) {
                        if constexpr (std::is_same_v<T,double>) 
                            tmp[i] = *reinterpret_cast<const uint64_t*>(&blk[beg+i]);
                        else 
                            tmp[i] = *reinterpret_cast<const uint32_t*>(&blk[beg+i]);
                    }
                    
                    ALPrdDict<T> D;
                    alprd_find_best<T>(tmp, len, D);
                    
                    // 记录调试信息
                    if (enable_diag && dbg_modes) {
                        uint64_t gid = vec_prefix[blockId] + (uint64_t)v;
                        dbg_modes[gid] = 1;  // ALPrd
                        if (dbg_e) dbg_e[gid] = 0xFF;
                        if (dbg_f) dbg_f[gid] = 0xFF;
                    }
                    
                    alprd_vector_write<T>(bw, tmp, len, D);
                }
            }
        }
        return;
    }


    // 并行处理（向量数较少时）
    __shared__ uint64_t sh_vec_bits[MAX_VECS_PER_BLOCK];
    __shared__ uint64_t sh_vec_offsets[MAX_VECS_PER_BLOCK + 1];
    __shared__ uint8_t sh_vec_e[MAX_VECS_PER_BLOCK];
    __shared__ uint8_t sh_vec_f[MAX_VECS_PER_BLOCK];
    __shared__ ALPrdDict<T> sh_vec_dict[MAX_VECS_PER_BLOCK];  // ALPrd字典
    
    // 第一步：并行计算每个向量的位数和参数
    for (int v = threadIdx.x; v < numVec; v += blockDim.x) {
        int beg = v * vectorSize;
        int rem = n - beg;
        int len = (vectorSize < rem ? vectorSize : rem);
        
        if (mode == CompressionMode::ALP) {
            // 直接对完整向量计算最优(e,f)
            uint8_t e, f;
            short bitw;
            long long FOR;
            int exc;
            vector_choose_from_k_combinations<T>(
                blk + beg, len,
                sh_k_combinations, sh_k_actual,
                e, f, bitw, FOR, exc
            );
            uint64_t bits = alp_vector_size_bits<T>(len, e, f, bitw, exc);
            sh_vec_bits[v] = bits;
            sh_vec_e[v] = e;
            sh_vec_f[v] = f;
        } else {
            // ALPrd模式
            uint64_t tmp[MAX_VEC];
            assert(len <= MAX_VEC);
            for (int i = 0; i < len; ++i) {
                if constexpr (std::is_same_v<T,double>) 
                    tmp[i] = *reinterpret_cast<const uint64_t*>(&blk[beg+i]);
                else 
                    tmp[i] = *reinterpret_cast<const uint32_t*>(&blk[beg+i]);
            }
            
            ALPrdDict<T> D;
            alprd_find_best<T>(tmp, len, D);
            
            // 统计异常
            int exc = 0;
            for (int i = 0; i < len; ++i) {
                uint32_t left = (uint32_t)((tmp[i] >> D.rightBW) & mask_lo(D.leftBW));
                bool inDict = false;
                for (int k = 0; k < DICT_SZ; ++k) {
                    if (D.dict[k] == left) { 
                        inDict = true; 
                        break; 
                    }
                }
                if (!inDict) exc++;
            }
            
            uint64_t bits = alprd_vector_size_bits<T>(len, D, exc);
            sh_vec_bits[v] = bits;
            sh_vec_dict[v] = D;  // 保存字典
        }
    }
    
    __syncthreads();
    
    // 第二步：计算前缀和（线程0执行串行扫描）
    if (threadIdx.x == 0) {
        sh_vec_offsets[0] = 8;  // 行组头占8位
        for (int v = 0; v < numVec; ++v) {
            sh_vec_offsets[v + 1] = sh_vec_offsets[v] + sh_vec_bits[v];
        }
    }
    __syncthreads();
    
    // 第三步：写行组头（线程0）
    if (threadIdx.x == 0) {
        BitWriter bw{out_bytes, bit_offsets[blockId]};
        bw.putN((uint64_t)numVec, 8);
    }
    __syncthreads();
    
    // 第四步：并行写入每个向量的压缩数据
    for (int v = threadIdx.x; v < numVec; v += blockDim.x) {
        int beg = v * vectorSize;
        int rem = n - beg;
        int len = (vectorSize < rem ? vectorSize : rem);
        
        // 每个线程独立的BitWriter，基于预计算的偏移
        BitWriter vec_bw{out_bytes, bit_offsets[blockId] + sh_vec_offsets[v]};
        
        if (mode == CompressionMode::ALP) {
            // 使用保存的(e,f)重新分析完整向量
            uint8_t e = sh_vec_e[v];
            uint8_t f = sh_vec_f[v];
            short bitw;
            long long FOR;
            int exc;
            alp_vector_analyze<T>(blk + beg, len, e, f, bitw, FOR, exc);
            
            // 记录调试信息
            if (enable_diag && dbg_modes) {
                uint64_t gid = vec_prefix[blockId] + (uint64_t)v;
                dbg_modes[gid] = 0;  // ALP
                if (dbg_e) dbg_e[gid] = e;
                if (dbg_f) dbg_f[gid] = f;
            }
            
            // 写入向量
            alp_vector_write<T>(vec_bw, blk + beg, len, e, f, bitw, FOR);
            
        } else { // ALPrd
            uint64_t tmp[MAX_VEC];
            assert(len <= MAX_VEC);
            for (int i = 0; i < len; ++i) {
                if constexpr (std::is_same_v<T,double>) 
                    tmp[i] = *reinterpret_cast<const uint64_t*>(&blk[beg+i]);
                else 
                    tmp[i] = *reinterpret_cast<const uint32_t*>(&blk[beg+i]);
            }
            
            // 记录调试信息
            if (enable_diag && dbg_modes) {
                uint64_t gid = vec_prefix[blockId] + (uint64_t)v;
                dbg_modes[gid] = 1;  // ALPrd
                if (dbg_e) dbg_e[gid] = 0xFF;
                if (dbg_f) dbg_f[gid] = 0xFF;
            }
            
            // 使用保存的字典
            alprd_vector_write<T>(vec_bw, tmp, len, sh_vec_dict[v]);
        }
    }
}
template<typename T>
static Compressed compress_impl(const T* h_data, size_t n, const Params& p){
    Compressed c;
    if (n==0) { c.vectorSize = p.vectorSize; return c; }

    // 切块：一个线程处理一个数据块
    const int V = p.vectorSize;
    const int B = p.blockSize>0 ? p.blockSize : int(n);
    const int numBlocks = int( (n + B - 1)/B );

    std::vector<uint64_t> h_starts(numBlocks), h_sizes(numBlocks);
    size_t pos=0;
    for(int i=0;i<numBlocks;++i){
        h_starts[i]=pos;
        uint64_t sz = std::min<uint64_t>(B, n-pos);
        h_sizes[i]  = sz;
        pos += sz;
    }

    // 诊断开关：环境变量 ALP_GPU_DIAG=1
    bool diag = (std::getenv("ALP_GPU_DIAG") != nullptr);

    // 上传输入与分块表
    T* d_data=nullptr; 
    hipMalloc(&d_data, n*sizeof(T));
    hipMemcpy(d_data, h_data, n*sizeof(T), hipMemcpyHostToDevice);

    uint64_t *d_starts=nullptr, *d_sizes=nullptr;
    hipMalloc(&d_starts, numBlocks*sizeof(uint64_t));
    hipMalloc(&d_sizes,  numBlocks*sizeof(uint64_t));
    hipMemcpy(d_starts, h_starts.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_sizes,  h_sizes.data(),  numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);

    // 第一阶段：测量大小（使用两级采样）
    uint64_t* d_bits=nullptr;  
    uint8_t* d_mode=nullptr;
    hipMalloc(&d_bits, numBlocks*sizeof(uint64_t));
    hipMalloc(&d_mode, numBlocks*sizeof(uint8_t));

    // 新kernel：使用两级采样测量大小
    dim3 grid1(numBlocks);
    dim3 block1(THREADS_PER_BLOCK);
    
    kernel_measure_with_sampling<T><<<grid1, block1>>>(
        d_data, d_starts, d_sizes, numBlocks, V, 
        d_bits, d_mode
    );
    hipDeviceSynchronize();

    std::vector<uint64_t> h_bits(numBlocks); 
    std::vector<uint8_t> h_mode(numBlocks);
    hipMemcpy(h_bits.data(), d_bits, numBlocks*sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_mode.data(), d_mode, numBlocks*sizeof(uint8_t), hipMemcpyDeviceToHost);

    // exclusive-scan 计算 bit 偏移（中间对齐）
    std::vector<uint64_t> h_off(numBlocks), padded_bits(numBlocks);
    uint64_t acc = 0;
    for (int i = 0; i < numBlocks; ++i) {
        h_off[i] = acc;
        uint64_t bits = h_bits[i];
        // 对除了最后一块之外，都向上补齐到 32bit 边界
        uint64_t pad = (i + 1 < numBlocks) ? ((32 - (bits & 31ULL)) & 31ULL) : 0ULL;
        padded_bits[i] = bits + pad;
        acc += padded_bits[i];
    }
    const uint64_t total_bits  = acc;
    const uint64_t total_bytes = (total_bits + 7) / 8;

    // 诊断：准备"全局向量 ID"前缀（block → global vec id）
    std::vector<uint64_t> h_vec_cnt(numBlocks), h_vec_prefix(numBlocks+1, 0);
    uint64_t total_vecs = 0;
    for (int i=0;i<numBlocks;++i){
        uint64_t cnt = (h_sizes[i] + (uint64_t)V - 1) / (uint64_t)V;
        h_vec_cnt[i] = cnt;
        h_vec_prefix[i+1] = h_vec_prefix[i] + cnt;
        total_vecs += cnt;
    }

    uint64_t* d_vec_prefix = nullptr;
    hipMalloc(&d_vec_prefix, sizeof(uint64_t)*(numBlocks+1));
    hipMemcpy(d_vec_prefix, h_vec_prefix.data(), sizeof(uint64_t)*(numBlocks+1), hipMemcpyHostToDevice);

    // 分配输出 buffer
    uint8_t* d_out=nullptr; 
    hipMalloc(&d_out, total_bytes);
    hipMemset(d_out, 0, total_bytes);
    
    uint64_t* d_off=nullptr; 
    hipMalloc(&d_off, numBlocks*sizeof(uint64_t));
    hipMemcpy(d_off, h_off.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);

    // 调试缓冲（可选）
    uint8_t *d_dbg_modes=nullptr, *d_dbg_e=nullptr, *d_dbg_f=nullptr;
    if (diag && total_vecs>0){
        hipMalloc(&d_dbg_modes, sizeof(uint8_t)*total_vecs);
        hipMalloc(&d_dbg_e,     sizeof(uint8_t)*total_vecs);
        hipMalloc(&d_dbg_f,     sizeof(uint8_t)*total_vecs);
        hipMemset(d_dbg_modes, 0xFF, sizeof(uint8_t)*total_vecs);
        hipMemset(d_dbg_e,     0xFF, sizeof(uint8_t)*total_vecs);
        hipMemset(d_dbg_f,     0xFF, sizeof(uint8_t)*total_vecs);
    }

    // 第二阶段：使用两级采样写入压缩数据
     kernel_emit_with_sampling<T><<<grid1, block1>>>(
        d_data, d_starts, d_sizes, d_off, d_mode,
        d_vec_prefix, d_dbg_modes, d_dbg_e, d_dbg_f,
        diag ? 1 : 0, numBlocks, V, d_out
    );
    hipDeviceSynchronize();

    // 拷回结果 & per-block 元信息
    c.data.resize(total_bytes);
    hipMemcpy(c.data.data(), d_out, total_bytes, hipMemcpyDeviceToHost);
    c.offsets = std::move(h_off);
    c.bit_sizes = std::move(h_bits);
    c.elem_counts.assign(h_sizes.begin(), h_sizes.end());
    c.vectorSize = V;

    // === 诊断打印（GPU 端）：模式分布 + 抽样 (e,f) ===
    if (diag && total_vecs>0){
        std::vector<uint8_t> dbg_modes(total_vecs, 0xFF), dbg_e(total_vecs, 0xFF), dbg_f(total_vecs, 0xFF);
        hipMemcpy(dbg_modes.data(), d_dbg_modes, sizeof(uint8_t)*total_vecs, hipMemcpyDeviceToHost);
        hipMemcpy(dbg_e.data(),     d_dbg_e,     sizeof(uint8_t)*total_vecs, hipMemcpyDeviceToHost);
        hipMemcpy(dbg_f.data(),     d_dbg_f,     sizeof(uint8_t)*total_vecs, hipMemcpyDeviceToHost);

        // 模式分布（向量粒度）
        uint64_t alp_cnt=0, alprd_cnt=0;
        for (uint64_t i=0;i<total_vecs;++i){
            if (dbg_modes[i]==0) ++alp_cnt;
            else if (dbg_modes[i]==1) ++alprd_cnt;
        }
        std::cout << "[GPU-Diag] Vector-mode distribution: ALP="<<alp_cnt
                  << ", ALPrd="<<alprd_cnt << ", totalVec="<< total_vecs << "\n";
    }

    // 清理
    if (d_dbg_modes) hipFree(d_dbg_modes);
    if (d_dbg_e)     hipFree(d_dbg_e);
    if (d_dbg_f)     hipFree(d_dbg_f);
    if (d_vec_prefix) hipFree(d_vec_prefix);

    hipFree(d_out); 
    hipFree(d_off);
    hipFree(d_mode); 
    hipFree(d_bits);
    hipFree(d_sizes); 
    hipFree(d_starts);
    hipFree(d_data);
    
    return c;
}
template<typename T>
static void decompress_impl(const Compressed& c, T* h_out, size_t n, const Params& p){
    if (n==0) return;
    const int numBlocks = (int)c.offsets.size();
    assert((size_t)numBlocks == c.elem_counts.size());

    // 上传压缩位流与 per-block 信息
    uint8_t* d_bytes=nullptr; hipMalloc(&d_bytes, c.data.size());
    hipMemcpy(d_bytes, c.data.data(), c.data.size(), hipMemcpyHostToDevice);

    uint64_t *d_boff=nullptr, *d_bsiz=nullptr, *d_ost=nullptr;
    hipMalloc(&d_boff, numBlocks*sizeof(uint64_t));
    hipMalloc(&d_bsiz, numBlocks*sizeof(uint64_t));
    hipMemcpy(d_boff, c.offsets.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_bsiz, c.bit_sizes.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);

    // 由 elem_counts 前缀和得到每块输出起点
    std::vector<uint64_t> h_outStarts(numBlocks);
    uint64_t acc=0; for(int i=0;i<numBlocks;++i){ h_outStarts[i]=acc; acc+=c.elem_counts[i]; }
    assert(acc == n && "elem_counts 总和必须等于输出元素数");

    hipMalloc(&d_ost, numBlocks*sizeof(uint64_t));
    hipMemcpy(d_ost, h_outStarts.data(), numBlocks*sizeof(uint64_t), hipMemcpyHostToDevice);

    T* d_out=nullptr; hipMalloc(&d_out, n*sizeof(T));

    dim3 gs(numBlocks), bs(1);
    kernel_decompress<T><<<gs,bs>>>(d_bytes, d_boff, d_bsiz, d_ost, p.vectorSize, d_out, numBlocks);
    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, n*sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_ost);
    hipFree(d_bsiz);
    hipFree(d_boff);
    hipFree(d_bytes);
}

// 显式实例化 API
Compressed compress_double(const double* data, size_t n, const Params& p){ return compress_impl<double>(data,n,p); }
Compressed compress_float (const float*  data, size_t n, const Params& p){ return compress_impl<float >(data,n,p); }
void decompress_double(const Compressed& c, double* out, size_t n, const Params& p){ decompress_impl<double>(c,out,n,p); }
void decompress_float (const Compressed& c, float*  out, size_t n, const Params& p){ decompress_impl<float >(c,out,n,p); }

} // namespace alp_gpu